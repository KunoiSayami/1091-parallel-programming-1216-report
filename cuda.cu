
#include <hip/hip_runtime.h>
#include <iostream>

typedef int data_type;

struct Matrix {
	int width;
	int height;
	data_type *elements;
	void setProp(int width_, int height_) {
		this->width = width_;
		this->height = height_;
	}
};

__device__ data_type getElement(Matrix *matrix, int row, int col)
{
	return matrix->elements[row * matrix->width + col];
}

__device__ void setElement(Matrix *matrix, int row, int col, int value)
{
	matrix->elements[row * matrix->width + col] = value;
}

__global__ void matMulKernel(Matrix *a, Matrix *b, Matrix *c)
{
	data_type value = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < a->width; i++)
	{
		value += getElement(a, row, i) * getElement(b, i, col);
	}
	setElement(c, row, col, value);
}

inline void printMatrix(Matrix * matrix) {
	for (int i = 0; i < matrix->width * matrix->height; i++) {
		std::cout << (i % matrix->width == 0 ? "\n" : " ") << matrix->elements[i];
	}
}

int main()
{
	int width, height;
	std::cin >> width >> height;

	Matrix *matrix_a, *matrix_b, *matrix_result;

	hipMallocManaged((void**)&matrix_a, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_b, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_result, sizeof(Matrix));
	size_t nBytes = width * height * sizeof(int);
	hipMallocManaged((void**)&matrix_a->elements, nBytes);
	hipMallocManaged((void**)&matrix_b->elements, nBytes);
	hipMallocManaged((void**)&matrix_result->elements, nBytes);

	matrix_a->setProp(width, height);
	matrix_b->setProp(width, height);
	matrix_result->setProp(width, height);

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_a->elements[i];

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_b->elements[i];

	dim3 blockSize(32, 32);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
				  (height + blockSize.y - 1) / blockSize.y);
	matMulKernel<<<gridSize, blockSize>>>(matrix_a, matrix_b, matrix_result);

	hipDeviceSynchronize();

	for (int i = 0; i < width * height; i++) {
		if (i > 0)
			std::cout << (i % width == 0 ? "\n" : " ");
		std::cout << matrix_result->elements[i];
	}
	std::cout << "\n";

	return 0;
}