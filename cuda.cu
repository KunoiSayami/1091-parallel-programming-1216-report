
#include <hip/hip_runtime.h>
#include <iostream>

struct Matrix
{
	int width;
	int height;
	int *elements;
	void setProp(int width_, int height_) {
		this->width = width_;
		this->height = height_;
	}
};

__device__ int getElement(Matrix *matrix, int row, int col)
{
	return matrix->elements[row * matrix->width + col];
}

__device__ void setElement(Matrix *matrix, int row, int col, int value)
{
	matrix->elements[row * matrix->width + col] = value;
}

__global__ void matMulKernel(Matrix *a, Matrix *b, Matrix *c)
{
	int value = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < a->width; ++i)
	{
		value += getElement(a, row, i) * getElement(b, i, col);
	}
	setElement(c, row, col, value);
}


int main()
{
	int width, height;
	std::cin >> width >> height;

	//std::cout << width << "*" << height << std::endl;
	Matrix *matrix_a, *matrix_b, *matrix_result;

	hipMallocManaged((void**)&matrix_a, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_b, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_result, sizeof(Matrix));
	size_t nBytes = width * height * sizeof(int);
	hipMallocManaged((void**)&matrix_a->elements, nBytes);
	hipMallocManaged((void**)&matrix_b->elements, nBytes);
	hipMallocManaged((void**)&matrix_result->elements, nBytes);

	matrix_a->setProp(width, height);
	matrix_b->setProp(width, height);
	matrix_result->setProp(width, height);

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_a->elements[i];

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_a->elements[i];

	dim3 blockSize(32, 32);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
				  (height + blockSize.y - 1) / blockSize.y);
	matMulKernel<<<gridSize, blockSize>>>(matrix_a, matrix_b, matrix_result);

	hipDeviceSynchronize();

	for (int i = 0; i < width * height; i++) {
		std::cout << (i % width == 0 ? "\n" : " ") << matrix_result->elements[i];
	}

	return 0;
}