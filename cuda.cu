
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

typedef float data_type;

struct Matrix {
	int width;
	int height;
	data_type *elements;
	void setProp(int width_, int height_) {
		this->width = width_;
		this->height = height_;
	}
};

__device__ data_type getElement(Matrix *matrix, int row, int col)
{
	return matrix->elements[row * matrix->width + col];
}

__device__ void setElement(Matrix *matrix, int row, int col, int value)
{
	matrix->elements[row * matrix->width + col] = value;
}

__global__ void matMulKernel(Matrix *a, Matrix *b, Matrix *c)
{
	data_type value = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i <= a->width; i++) {
		value += getElement(a, row, i) * getElement(b, i, col);
	}
	setElement(c, row, col, value);
}

inline void printMatrix(Matrix * matrix) {
	for (int i = 0; i < matrix->width * matrix->height; i++) {
		std::cout << (i % matrix->width == 0 ? "\n" : " ") << matrix->elements[i];
	}
}

int gcd(int a, int b) {
	return b == 0? a : gcd(b, a%b);
}

int main()
{
	int width, height;
	std::cin >> width >> height;

	Matrix *matrix_a, *matrix_b, *matrix_result;

	hipMallocManaged((void**)&matrix_a, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_b, sizeof(Matrix));
	hipMallocManaged((void**)&matrix_result, sizeof(Matrix));
	size_t nBytes = width * height * sizeof(data_type);
	hipMallocManaged((void**)&matrix_a->elements, nBytes);
	hipMallocManaged((void**)&matrix_b->elements, nBytes);
	hipMallocManaged((void**)&matrix_result->elements, nBytes);

	matrix_a->setProp(width, height);
	matrix_b->setProp(width, height);
	matrix_result->setProp(width, height);

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_a->elements[i];

	for (int i = 0; i < width * height; i++)
		std::cin >> matrix_b->elements[i];

	dim3 blockSize(gcd(width, 32), gcd(width, 32));
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
				  (height + blockSize.y - 1) / blockSize.y);

	high_resolution_clock::time_point start_time = high_resolution_clock::now();
	matMulKernel<<<gridSize, blockSize>>>(matrix_a, matrix_b, matrix_result);

	std::cerr << "Time spend: " << duration_cast<duration<double>>(high_resolution_clock::now() - start_time).count() << "\n";

	hipDeviceSynchronize();

	for (int i = 0; i < width * height; i++) {
		if (i > 0)
			std::cout << (i % width == 0 ? "\n" : " ");
		std::cout << (int)matrix_result->elements[i];
	}
	std::cout << "\n";

	hipFree(matrix_a);
	hipFree(matrix_b);
	hipFree(matrix_result);
	return 0;
}